
#include <hip/hip_runtime.h>

#define NOW(time, i) (2 * (i) + time)
#define FUTURE(time, i) (2 * (i) + (1 - time))

extern "C"
__global__ void move(int n, int time, int *a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i > 2) && (2 * i + 1 < n)) {
        char p = a[NOW(time, i - 1)];
        char q = a[NOW(time, i)];
        char r = a[NOW(time, i + 1)];
        a[FUTURE(time, i)] = ((!p && q && r) || (p && !q && !r) || (p && !q && r) || (p && q && r));
    }
}